
#include <hip/hip_runtime.h>

#define bottom_data(n,c,h,w) bottom_data[(n)*H*W*C+(c)*H*W+(h)*W+(w)]
#define top_data(n,c,h,w) top_data[(n)*OC*OH*OW+(c)*OH*OW+(h)*OW+(w)]
#define kernel(n,c,h,w) kernel[(n)*C*FW*FH+(c)*FW*FH+(h)*FW+(w)]
__global__ void DPUDirectConv(
        int N, int C, int H, int W,float *bottom_data,
        int OC1,int C1,int FH,int FW,float *kernel,
        int N1,int OC, int OH, int OW, float *top_data,
        int SH, int SW, int PH, int PW)
{
    int init_pw = (threadIdx.x) * 7; 
    int init_ph = (blockIdx.y * blockDim.y + threadIdx.y) * 7 ;
    int init_c = (blockIdx.z * blockDim.z + threadIdx.z)* 1;
    #pragma SIMD (n)
    #pragma unroll 
    for(int n = 0; n < N; ++n){
        #pragma unroll
        for (int c = 0; c < 1; ++c) {
            #pragma unroll
            for (int ph = 0; ph < 7; ++ph) {
                #pragma unroll
                for (int pw = 0; pw < 7; ++pw) {
                    if((ph+init_ph) < OH &&  (pw+init_pw) < OW && (init_c+c) < OC){
                        int hs = (ph+init_ph) * SH -PH;
                        int ws = (pw+init_pw) * SW -PW;
                        int hend = min(hs + FH, H);
                        int wend = min(ws + FW, W);
                        int hstart = max(hs, 0);
                        int wstart = max(ws, 0);
                        float sum = 0.0f;
                        //cal
                        #pragma reduction (sum,yc,+,64)
                        for(int yc=0;yc < C; ++yc){
                            for (int h = hstart-hs; h < hend-hs; ++h) {
                                for (int w = wstart-ws; w < wend-ws; ++w) { 
                                    sum += bottom_data(n,yc,h+hs,w+ws) * kernel(init_c+c,yc,h,w);            
                                }
                            }
                        }
                        //ST
                        top_data(n,init_c+c,ph+init_ph,pw+init_pw) = sum;   
                    }               
                }
            }
        }
    }
}

int main(){

    int N = 32 ;            //input & output nums
    int C = 256;           //input_channel
    int H= 13;             //input_height
    int W= 13;              //input 
    int OC = 32;   //output_channel
    int OH = 13;     //OH = (H + PH * 2 - FH)/SH + 1
    int OW = 13;      //OW = (W + PW * 2 - FW)/SW + 1
    int SH = 1;
    int SW = 1;
    int PH = 1;
    int PW = 1;
    int FH = 3;
    int FW = 3;
    
    float *device_bottom_data;
    float *device_top_data;
    float *device_kernel;
    
    dim3 grid(1,1,2);//width,height,(channel*num)
    dim3 block(2,2,16);//width,hegiht,channel

   
    DPUDirectConv<<<grid,block>>>(N, C, H, W,device_bottom_data, OC,C,FH,FW,device_kernel,N,OC,OH,OW,device_top_data,SH, SW, PH, PW);

    return 0;
}
