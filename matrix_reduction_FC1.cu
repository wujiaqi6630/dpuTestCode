
#include <hip/hip_runtime.h>
// C = alpha * A * B + beta * C
__global__ void MatMulKernelAB(const int M,const int K,float *A,
								const int K1, const int N,float *B, 
								const int M1, const int N1,float *C,
								const float alpha,const float beta)
{
	// Each thread computes at most (UNROLL_X * UNROLL_Y) elements of C
	// by accumulating results into Cvalue
	int row = threadIdx.y*2;
	int col = threadIdx.x*128;
	#pragma unroll 
	for(int j = 0; j < 2; ++j){
		#pragma SIMD (i)
		#pragma unroll
		for(int i = 0; i < 128; ++i){
			float Cvalue = C[(row+j)*N+(col+i)];
			for(int e = 0; e < K; ++e){
				#pragma reduction (Cvalue,e,+,1024)
				Cvalue += A[(row+j)*K + e] * B[e*N + (col+i)];
			}
			//C[row][col] = alpha * Cvalue + beta * C[row][col];
			C[(row+j)*N+(col+i)] = Cvalue  ;
		}
	}
}
int main(int argc, char const *argv[])
{
	int M = 32;
	int N = 512;
	int K = 9216;

	float *A;
	float *B;
	float *C;

	dim3 blockDim(4,16,1);
	dim3 gridDim(1,1,1);

	MatMulKernelAB<<<gridDim,blockDim>>>(M,K,A,K,N,B,M,N,C,1.0,0.0);
	return 0;
}
