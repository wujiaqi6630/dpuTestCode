
#include <hip/hip_runtime.h>

#define bottom_data(n,c,h,w) bottom_data[(n)*H*W*C+(c)*H*W+(h)*W+(w)]
#define top_data(n,c,h,w) top_data[(n)*OC*OH*OW+(c)*OH*OW+(h)*OW+(w)]
#define kernel(n,c,h,w) kernel[(n)*C*FW*FH+(c)*FW*FH+(h)*FW+(w)]
__global__ void DPUPooling(
        int N, int C, int H, int W,float *bottom_data,
        int N1,int OC, int OH, int OW, float *top_data,
        int SH, int SW, int PH, int PW, int FH, int FW)
{
    int init_pw = (blockIdx.x * blockDim.x + threadIdx.x) * 7; 
    int init_ph = (blockIdx.y * blockDim.y + threadIdx.y) * 7 ;
    int init_c = (blockIdx.z * blockDim.z + threadIdx.z)* 1 ;
    #pragma SIMD (n)
    #pragma unroll 
    for(int n = 0; n < N; ++n){
        #pragma unroll
        for (int c = 0; c < 1; ++c) {
            #pragma unroll
            for (int ph = 0; ph < 7; ++ph) {
                #pragma unroll
                for (int pw = 0; pw < 7; ++pw) {
                    if((ph+init_ph) < OH &&  (pw+init_pw) < OW && (init_c+c) < OC){
                        int hs = (ph+init_ph) * SH -PH;
                        int ws = (pw+init_pw) * SW - PW;
                        int hend = min(hs + FH, H);
                        int wend = min(ws + FW, W);
                        int hstart = max(hs, 0);
                        int wstart = max(ws, 0);
                        float sum = 0.0f;
                        //cal
                        
                            for (int h = hstart-hs; h < hend-hs; ++h) {
                                for (int w = wstart-ws; w < wend-ws; ++w) { 
                                    sum += bottom_data(n,init_c+c,h+hs,w+ws) ;            
                                }
                            }
                        //ST
                        top_data(n,init_c+c,ph+init_ph,pw+init_pw) = sum /4;   
                    }               
                }
            }
        }
    }
}

int main(){

    int N = 32 ;            //input & output nums
    int C = 96;           //input_channel
    int H= 55;             //input_height
    int W= 55;              //input 
    int OC = 96;   //output_channel
    int OH = 27;     //OH = (H + PH * 2 - FH)/SH + 1
    int OW = 27;      //OW = (W + PW * 2 - FW)/SW + 1
    int SH = 2;
    int SW = 2;
    int PH = 0;
    int PW = 0;
    int FH = 3;
    int FW = 3;
    
    float *device_bottom_data;
    float *device_top_data;
    
    dim3 grid(1,1,24);//width,height,(channel*num)
    dim3 block(4,4,4);//width,hegiht,channel

   
    DPUPooling<<<grid,block>>>(N, C, H, W,device_bottom_data, N,OC,OH,OW,device_top_data,SH, SW, PH, PW, FH, FW);

    return 0;
}
